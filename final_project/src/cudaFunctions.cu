#include "hip/hip_runtime.h"
//Sapir Madmon 209010230

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cudaFunctions.h"


//the func called from host(CPU) and executed on device (GPU) 
__global__ void calcBestScoreViaCuda(double *d_matW, char *seq1, char *seq2, int offSet, int numSeq2, double *d_arrScore, int lenSeq2) 
{
	int index_seq1, index_seq2;
	double score = 0;
	int tid = blockDim.x * blockIdx.x + threadIdx.x; //each tid is index of mutant
	
	tid++;	//should be increased by 1 (because the mutant index starts at position 1)

	if (tid > lenSeq2)
		return;

	for (index_seq2 = 0; index_seq2 < lenSeq2; index_seq2++) 
	{
		index_seq1 = index_seq2 + offSet;
		if (index_seq2 >= tid)
			index_seq1++;
		
		score += d_matW[NUM_ABC * (seq2[index_seq2] - 'A') + (seq1[index_seq1] - 'A')];
	}
	
	tid--; 
	d_arrScore[tid] = score;
}


double* computeArrScoreOnGPU(double *d_matW, char *seq1, char *seq2, int offSet, int numSeq2, int seq2Len) 
{	
	double *d_arrScore;
	size_t size_seq2Len = seq2Len * sizeof(double);
	
	//Allocate memory on GPU to copy the data from the host
	hipError_t(hipMalloc((void**) &d_arrScore, size_seq2Len), "Failed to allocate device memory arrScore - %s\n");

	double *arrScore = (double*) calloc(seq2Len, sizeof(double));  //arr score for all results 
	
	int threadsPerBlock = 256;	//number of threads per block
	int blocksPerGrid = (seq2Len + threadsPerBlock - 1) / threadsPerBlock; //compute number of blocks per grid  

	//kernel calc the best score and create array of result  
	calcBestScoreViaCuda<<<blocksPerGrid, threadsPerBlock>>>(d_matW, seq1, seq2, offSet, numSeq2, d_arrScore, seq2Len);
	
	//Copy the result (arrScore) from GPU to the host memory.
	hipMemcpy(arrScore, d_arrScore, size_seq2Len, hipMemcpyDeviceToHost);
	hipError_t(hipFree(d_arrScore), "Failed to free device arrScore - %s\n");

	return arrScore;
}


//Copy data from host to the GPU memory
double* allocatedWeightMat(double matW[NUM_ABC][NUM_ABC]) 
{
	double *d_matW = NULL;
	size_t size_matW = NUM_ABC * NUM_ABC * sizeof(double);
	
	//Allocate memory on GPU to copy the data from the host
	hipError_t(hipMalloc((void**) &d_matW, size_matW), "Failed to copy WeightMat from host to device - %s\n");
	hipMemcpy(d_matW, matW, size_matW, hipMemcpyHostToDevice); //Copy matW from host to the GPU memory

	return d_matW;
}


//Copy data from host to the GPU memory
char* allocatedSeq1OnGPU(char *seq1)
{
	char *d_seq1 = NULL;
	size_t size_seq1 = (strlen(seq1)) * sizeof(char);
	
	//Allocate memory on GPU to copy the data from the host
	hipError_t(hipMalloc((void**) &d_seq1, size_seq1), "Failed to copy seq1 from host to device - %s\n");
	hipMemcpy(d_seq1, seq1, size_seq1, hipMemcpyHostToDevice); //Copy seq1 from host to the GPU memory

	return d_seq1;
}


//Copy data from host to the GPU memory
char* allocatedSeq2OnGPU(char *seq2) 
{
	char *d_seq2 = NULL;
	size_t size_seq2 = (strlen(seq2)) * sizeof(char);
	
	//Allocate memory on GPU to copy the data from the host
	hipError_t(hipMalloc((void**) &d_seq2, size_seq2), "Failed to copy seq2 from host to device - %s\n");
	hipMemcpy(d_seq2, seq2, size_seq2, hipMemcpyHostToDevice); //Copy seq2 from host to the GPU memory

	return d_seq2;
}


//Free allocated memory on GPU
void freeAllocated_matW(double *d_matW) 
{
	hipError_t(hipFree(d_matW), "Failed to free device matW - %s\n");
}

//Free allocated memory on GPU
void freeAllocated_seq1(char *d_seq1) 
{
	hipError_t(hipFree(d_seq1), "Failed to free device seq1 - %s\n");
}

//Free allocated memory on GPU
void freeAllocated_seq2(char *d_seq2) 
{
	hipError_t(hipFree(d_seq2), "Failed to free device seq2 - %s\n");
}


void hipError_t(hipError_t err, const char *messageError) 
{
	if (err != hipSuccess) 
	{
		fprintf(stderr, messageError, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
